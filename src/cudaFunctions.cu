#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "myProto.h"
#include "omp.h"

#define THREADS_PER_BLOCK 1024

int k,n;

__device__ const int PRIME_NUMBERS[] = {
	2,3,5,7,11,13,17,19,23,29,31,37,41,43,47,53,59,61,67,71,73,79,83,89,97
};

__device__ int gpu_pair(char sequence1_char, char sequence2_char, int* group, int group_size)
{
	int offset = 'A';
	int mul = PRIME_NUMBERS[sequence1_char - offset] * PRIME_NUMBERS[sequence2_char - offset];
	for(int i = 0 ; i < group_size ; i++){
		if(group[i]%mul == 0)
			return 1;
	}
	return 0;
}

__global__ void gpu_calc_str_weight(const char* sequence1, int sequence1_length, char* sequence2, int sequence2_length,
		int max_k, int max_n, float* results, const weights_t* weights, int* conservative_group,
		int conservative_group_size, int* semi_conservative_group, int semi_conservative_group_size, int max_threads)
{
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;
	//kill over threads
	if(threadId > max_threads -1)
		return;

	int k = threadId/max_n;
	int n = threadId%max_n;
	if(k==0)
		return;
	for(int i = 0 ; i < sequence2_length ; i++){
		int j = 0;
		if(i>= k)
			j = 1;
		if(i == k){
			results[k + n*max_k] -= weights->w4;
		}
		if(sequence1[j+n+i] == sequence2[i]){
			results[k + n*max_k] += weights->w1;
		}else if(gpu_pair(sequence1[j+n+i],sequence2[i],conservative_group,conservative_group_size)){
			results[k + n*max_k] -= weights->w2;
		}else if(gpu_pair(sequence1[j+n+i],sequence2[i],semi_conservative_group,semi_conservative_group_size)){
			results[k + n*max_k] -= weights->w3;
		}else{
			results[k + n*max_k] -= weights->w4;
		}
	}
}

void find_best_k_n(float* results, int max_k, int max_n, int* best_k, int* best_n)
{
//The minimum value of float.
	float best_weight = -10e38;
#pragma omp parallel for collapse(2)
	for(int n = 0 ; n < max_n ; n++){
		for(int k = 0 ; k < max_k ; k++){
#pragma omp critical
{
	// k is the col n is the rows
			if(results[k + n*max_k] > best_weight){
				best_weight = results[k + n*max_k];
				*best_k = k;
				*best_n = n;
			}
}
		}
	}
}

void get_best_results_with_CUDA(const char* sequence1, int sequence1_length, char* sequence2, int sequence2_length,
		const weights_t* weights, int* results,
		int* conservative_group, int conservative_group_size, int* semi_conservative_group,
		int semi_conservative_group_size)
{
// Allocate space for sequence1 on GPU
	char* gpu_sequence1;
	hipMalloc(&gpu_sequence1,sequence1_length+1);
	
// Allocate space for sequence2 on GPU
	char* gpu_sequence2;
	hipMalloc(&gpu_sequence2,sequence2_length+1);

// Copy sequence1 to GPU
	hipMemcpy(gpu_sequence1,sequence1,sequence1_length+1,hipMemcpyHostToDevice);
	
// Copy sequence1 to GPU
	hipMemcpy(gpu_sequence2,sequence2,sequence2_length+1,hipMemcpyHostToDevice);

// Allocate space for 4 weights on GPU
	weights_t* gpu_weights;
	hipMalloc(&gpu_weights,sizeof(weights_t));
	
	// Copy weights to GPU
	hipMemcpy(gpu_weights,weights,sizeof(weights_t),hipMemcpyHostToDevice);

// Allocate space for conservative group on GPU
	int* gpu_conservative_group;
	hipMalloc(&gpu_conservative_group,sizeof(int)*conservative_group_size);
	// Copy conservative group to gpu
	hipMemcpy(gpu_conservative_group,conservative_group,conservative_group_size * sizeof(int),hipMemcpyHostToDevice);

// Allocate space for semi conservative group on GPU
	int* gpu_semi_conservative_group;
	hipMalloc(&gpu_semi_conservative_group,semi_conservative_group_size*sizeof(int));
	// Copy semi conservative group to gpu
	hipMemcpy(gpu_semi_conservative_group,semi_conservative_group,semi_conservative_group_size * sizeof(int),hipMemcpyHostToDevice);

	dim3 grid, block;
	// Max size of hypen options
	int max_k = sequence2_length+1;
	// Max size of offset options
	int max_n = sequence1_length - sequence2_length;
//	int num_of_blocks = (sequence2_length*max_k*max_n/THREADS_PER_BLOCK)+1;

	// Max size of blocks options
	int num_of_blocks = (max_k*max_n/THREADS_PER_BLOCK)+1;
	// Set y grid axis
	grid.y = 1;
	// Set z grid axis 
	grid.z = 1;
	// Set x grid axis 
	grid.x = num_of_blocks;
	
	// Set y block axis to max
	block.y = 1;
	// Set z block axis to max
	block.z = 1;
	// Set x block axis to max
	block.x = THREADS_PER_BLOCK;
	// Allocate space for results on GPU and set to zero (why zero here ??? i did async)
	float* gpu_results;
	hipMalloc(&gpu_results,max_k*max_n*sizeof(float));
	hipMemsetAsync(gpu_results,0,max_k*max_n*sizeof(float));
	gpu_calc_str_weight<<<grid,block>>>(gpu_sequence1,sequence1_length,gpu_sequence2,sequence2_length,max_k,max_n,
			gpu_results,gpu_weights,gpu_conservative_group,conservative_group_size,gpu_semi_conservative_group,
			semi_conservative_group_size,max_k*max_n);
	//Cuda barrier
	hipDeviceSynchronize();
	// Allocate space for results on CPU
	float* cpu_results = (float*)calloc(max_k*max_n,sizeof(float));
	// Copy result from gpu to cpu
	hipMemcpy(cpu_results,gpu_results,max_k*max_n*sizeof(float),hipMemcpyDeviceToHost);
	find_best_k_n(cpu_results,max_k,max_n,&k,&n);
	//Write best k in first cell in result CPU array
	results[0] = k;
	//Write best n in second cell in result CPU array
	results[1] = n;

	//Free GPU resource
	hipFree(gpu_sequence1);
	hipFree(gpu_sequence2);
	hipFree(gpu_weights);
	hipFree(gpu_conservative_group);
	hipFree(gpu_semi_conservative_group);
	hipFree(gpu_results);
	//Free CPU resource 
	free(cpu_results);
}
